#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <assert.h>
#include <stdio.h>

const int SIZE = 32;
const int MATRIX_SIZE = 1000;

hipError_t addWithCuda(float *c, const float *a, const float *b);

__global__ void addKernel(float *c, const float *a, const float *b)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < MATRIX_SIZE && col < MATRIX_SIZE)
	{
		auto v = 0.f;
		for (auto x = 0; x < MATRIX_SIZE; x++)
		{
			v += a[row * MATRIX_SIZE + x] * b[x * MATRIX_SIZE + col];
		}

		c[row * MATRIX_SIZE + col] = v;
	}
}

int main()
{
	auto k = MATRIX_SIZE;
	auto l = MATRIX_SIZE;
	auto m = MATRIX_SIZE;

	auto a = new float[k * l];
	auto b = new float[l * m];
	auto dc = new float[k * m]; // for device
	auto hc = new float[k * m]; // for host

	for (auto i = 0; i < k*l; i++)
	{
		a[i] = 0.01f * i;
	}
	for (auto i = 0; i < l*m; i++)
	{
		b[i] = 0.01f * i;
	}

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(dc, a, b);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	std::chrono::high_resolution_clock::time_point start, stop;

	start = std::chrono::high_resolution_clock::now();
	for (auto i = 0; i < MATRIX_SIZE; i++)
	{
		for (auto j = 0; j < MATRIX_SIZE; j++)
		{
			auto v = 0.f;
			for (auto t = 0; t < MATRIX_SIZE; t++)
			{
				v += a[i * MATRIX_SIZE + t] * b[t * MATRIX_SIZE + j];
			}

			hc[i * MATRIX_SIZE + j] = v;
		}
	}
	stop = std::chrono::high_resolution_clock::now();

	printf("No CUDA is\n");
	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	printf("\ttime = {%lld}\n", ms.count());

	// Check result
	for (auto i = 0; i < k * m; i++)
	{
		assert(hc[i] == dc[i]);
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, const float *a, const float *b)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	std::chrono::high_resolution_clock::time_point start, stop;
	start = std::chrono::high_resolution_clock::now();

	// Launch a kernel on the GPU with one thread for each element.
	dim3 threadsPerBlock(32, 32); // 32 x 32 threads / per block
	dim3 numBlocks(256, 256);     // 256 x 256 blocks / per grid

	printf("threadsPerBlock.X =%d, threadsPerBlock.y = %d\n", threadsPerBlock.x, threadsPerBlock.y);
	printf("numBlocks.X =%d, numBlocks.y = %d\n", numBlocks.x, numBlocks.y);

	addKernel << <numBlocks, threadsPerBlock >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	stop = std::chrono::high_resolution_clock::now();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	printf("CUDA is\n");

	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	printf("\ttime = {%lld}\n", ms.count());

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}