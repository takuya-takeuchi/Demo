
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_cuda()
{
    printf("Hello from CUDA kernel!\n");
}

#define BLOCK_SIZE 1
#define THREAD_PER_BLOCK 8

int main()
{
    printf("Start!\n");

    hello_cuda<<<BLOCK_SIZE, THREAD_PER_BLOCK>>>();
    hipDeviceSynchronize();

    printf("End!\n");
    
    return 0;
}