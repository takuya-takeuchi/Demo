
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_cuda()
{
    printf("Hello from CUDA kernel!\n");
}

int main()
{
    printf("Start!\n");

    hello_cuda<<<2, 4>>>();
    hipDeviceSynchronize();

    return 0;
}